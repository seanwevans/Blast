/*
blast.cu

  Unified high-throughput SQLite table dumper

  Build
    nvcc -O3 -Xcompiler -fopenmp -march=native -arch=sm_86 blast.cu -o blast

  Usage
    blast [--cuda] [--simd|--nosimd] [--table <name>] input.db output.csv
*/

#include <fcntl.h>
#include <omp.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <unistd.h>

#if defined(__HIPCC__)
#define HD __host__ __device__
#else
#define HD
#endif

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif
#ifdef __AVX2__
#include <immintrin.h>
#endif

#define PAGE_LEAF 13
#define PAGE_INTERIOR 5
#define MAX_CELLS 512
#define MAX_PATH_DEPTH 32

typedef struct {
  uint64_t row_id;    // synthetic id (page<<32 | cell)
  uint64_t rowid_val; // true SQLite rowid
  uint32_t page;
  uint16_t offset;
  uint16_t length;
} RecordTask;

typedef struct {
  uint32_t *pages;
  int count;
  int cap;
} PageList;

static void pagelist_init(PageList *pl) {
  pl->cap = 128;
  pl->count = 0;
  pl->pages = (uint32_t *)malloc(pl->cap * sizeof(uint32_t));
}

static void pagelist_push(PageList *pl, uint32_t page) {
  if (pl->count >= pl->cap) {
    pl->cap *= 2;
    pl->pages = (uint32_t *)realloc(pl->pages, pl->cap * sizeof(uint32_t));
  }
  pl->pages[pl->count++] = page;
}

static uint8_t *mmap_file(const char *path, size_t *sz) {
  int fd = open(path, O_RDONLY);
  if (fd < 0) {
    perror("open");
    exit(1);
  }
  struct stat st;
  if (fstat(fd, &st)) {
    perror("fstat");
    exit(1);
  }
  *sz = (size_t)st.st_size;
  uint8_t *p = (uint8_t *)mmap(NULL, *sz, PROT_READ, MAP_PRIVATE, fd, 0);
  if (p == MAP_FAILED) {
    perror("mmap");
    exit(1);
  }
  close(fd);
  return p;
}

static void *mmap_outfile(const char *path, size_t sz, int *fd_out) {
  int fd = open(path, O_RDWR | O_CREAT | O_TRUNC, 0644);
  if (fd < 0) {
    perror("open out");
    exit(1);
  }
  (void)posix_fallocate(fd, 0, (off_t)sz);
  void *m = mmap(NULL, sz, PROT_WRITE, MAP_SHARED, fd, 0);
  if (m == MAP_FAILED) {
    perror("mmap out");
    exit(1);
  }
  *fd_out = fd;
  return m;
}

static size_t detect_page_size(const uint8_t *db) {
  uint16_t ps = (uint16_t)((db[16] << 8) | db[17]);
  return ps == 1 ? 65536u : (size_t)ps;
}

static inline char *u64toa(uint64_t val, char *out) {
  char tmp[32];
  int i = 31;
  tmp[i--] = '\0';
  if (!val)
    tmp[i--] = '0';
  else
    while (val) {
      tmp[i--] = (char)('0' + (val % 10));
      val /= 10;
    }
  size_t len = (size_t)(31 - i);
  memcpy(out, tmp + i + 1, len - 1);
  return out + (len - 1);
}

static inline char *dtoa(double val, char *out) {
  char buf[32];
  int len = snprintf(buf, sizeof(buf), "%.17g", val);
  if (len < 0)
    len = 0;
  memcpy(out, buf, (size_t)len);
  return out + len;
}

static inline char *write_csv_text(const uint8_t *data, size_t len, char *out) {
  *out++ = '"';
  for (size_t i = 0; i < len; i++) {
    unsigned char c = data[i];
    if (c == '"') {
      *out++ = '"';
      *out++ = '"';
    } else {
      *out++ = (char)c;
    }
  }
  *out++ = '"';
  return out;
}

static inline char *write_blob_hex(const uint8_t *data, size_t len, char *out) {
  static const char hex[] = "0123456789ABCDEF";
  *out++ = 'X';
  *out++ = '\'';
  for (size_t i = 0; i < len; i++) {
    uint8_t byte = data[i];
    *out++ = hex[byte >> 4];
    *out++ = hex[byte & 0xF];
  }
  *out++ = '\'';
  return out;
}

HD static inline uint64_t read_varint_scalar(const uint8_t *p, int *len) {
  uint64_t v = 0;
  for (int i = 0; i < 9; i++) {
    uint8_t b = p[i];
    v = (v << 7) | (b & 0x7F);
    if (!(b & 0x80)) {
      *len = i + 1;
      return v;
    }
  }
  *len = 9;
  return v;
}

#ifdef __AVX2__
static inline uint64_t read_varint_avx2(const uint8_t *p, int *len) {
  __m128i v = _mm_loadu_si128((const __m128i *)p);
  __m128i mask = _mm_cmplt_epi8(v, _mm_set1_epi8(0x80));
  int m = _mm_movemask_epi8(mask);
  int idx = __builtin_ctz(m | 0x10000);
  *len = idx + 1;
  uint64_t x = 0;
  for (int i = 0; i <= idx; i++)
    x = (x << 7) | (p[i] & 0x7F);
  return x;
}
#endif

static int use_simd_runtime = 0;

static inline uint64_t read_varint_auto(const uint8_t *p, int *len) {
#if defined(__AVX2__)
  if (use_simd_runtime)
    return read_varint_avx2(p, len);
  else
    return read_varint_scalar(p, len);
#else
  (void)use_simd_runtime;
  return read_varint_scalar(p, len);
#endif
}

static char *decode_record(const uint8_t *payload, size_t payload_len,
                           uint64_t rowid_val, char *out) {
  int hdr_varlen = 0;
  uint64_t header_size = read_varint_auto(payload, &hdr_varlen);
  size_t header_end = (size_t)header_size;
  size_t header_pos = (size_t)hdr_varlen;
  size_t data_pos = header_end;
  int field_index = 0;

  while (header_pos < header_end && data_pos <= payload_len) {
    int vlen = 0;
    uint64_t serial_type = read_varint_auto(payload + header_pos, &vlen);
    header_pos += (size_t)vlen;
    if (field_index++ > 0)
      *out++ = ',';

    if (serial_type == 0) {
      if (field_index == 1)
        out = u64toa(rowid_val, out);
    } else if (serial_type == 7) {
      if (data_pos + 8 <= payload_len) {
        uint64_t bits = 0;
        for (int b = 0; b < 8; b++)
          bits = (bits << 8) | payload[data_pos + b];
        data_pos += 8;
        union {
          uint64_t u;
          double d;
        } conv;
        conv.u = bits;
        out = dtoa(conv.d, out);
      } else
        *out++ = '?';
    } else if (serial_type >= 1 && serial_type <= 6) {
      static const int blut[7] = {0, 1, 2, 3, 4, 6, 8};
      int blen = blut[serial_type];
      if (data_pos + (size_t)blen > payload_len)
        blen = (int)(payload_len - data_pos);
      uint64_t val = 0;
      for (int b = 0; b < blen; b++)
        val = (val << 8) | payload[data_pos + b];
      data_pos += (size_t)blen;
      out = u64toa(val, out);
    } else if (serial_type == 8) {
      *out++ = '0';
    } else if (serial_type == 9) {
      *out++ = '1';
    } else if (serial_type >= 12 && (serial_type % 2 == 0)) {
      size_t blen = (size_t)((serial_type - 12) / 2);
      if (data_pos + blen > payload_len)
        blen = payload_len - data_pos;
      out = write_blob_hex(payload + data_pos, blen, out);
      data_pos += blen;
    } else if ((serial_type >= 13) && (serial_type % 2 == 1)) {
      size_t tlen = (size_t)((serial_type - 13) / 2);
      if (data_pos + tlen > payload_len)
        tlen = payload_len - data_pos;
      out = write_csv_text(payload + data_pos, tlen, out);
      data_pos += tlen;
    } else
      *out++ = '?';
  }
  *out++ = '\n';
  return out;
}

#ifdef __HIPCC__
__global__ void gpu_scan_pages(const uint8_t *db, size_t page_sz,
                               size_t n_pages, RecordTask *out_tasks,
                               int *total) {
  int page_id = blockIdx.x;
  if (page_id >= (int)n_pages)
    return;
  const uint8_t *page = db + page_id * page_sz;
  if (page[0] != PAGE_LEAF)
    return;
  int nc = (page[3] << 8) | page[4];
  int base = atomicAdd(total, nc);
  for (int c = threadIdx.x; c < nc; c += blockDim.x) {
    uint16_t off = (page[8 + 2 * c] << 8) | page[8 + 2 * c + 1];
    int vlen1;
    uint64_t pay = read_varint_scalar(page + off, &vlen1);
    int vlen2;
    uint64_t rowid_val = read_varint_scalar(page + off + vlen1, &vlen2);
    RecordTask t;
    t.row_id = ((uint64_t)page_id << 32) | c;
    t.rowid_val = rowid_val;
    t.page = page_id;
    t.offset = (uint16_t)(off + vlen1 + vlen2);
    t.length = (uint16_t)pay;
    out_tasks[base + c] = t;
  }
}
#endif

static void collect_table_pages(const uint8_t *db, size_t db_sz, size_t page_sz,
                                uint32_t page_no, PageList *out) {
  if (page_no == 0 || (size_t)page_no * page_sz >= db_sz)
    return;
  const uint8_t *page = db + (size_t)(page_no - 1) * page_sz;
  uint8_t type = page[0];
  if (type == PAGE_LEAF) {
    pagelist_push(out, page_no - 1);
    return;
  }
  if (type != PAGE_INTERIOR)
    return;
  int nc = (page[3] << 8) | page[4];
  for (int i = 0; i < nc; i++) {
    uint16_t off = (page[12 + 2 * i] << 8) | page[12 + 2 * i + 1];
    uint32_t child = (page[off] << 24) | (page[off + 1] << 16) |
                     (page[off + 2] << 8) | (page[off + 3]);
    collect_table_pages(db, db_sz, page_sz, child, out);
  }
  uint32_t right =
      (page[8] << 24) | (page[9] << 16) | (page[10] << 8) | (page[11]);
  collect_table_pages(db, db_sz, page_sz, right, out);
}

static uint32_t find_table_rootpage_page(const uint8_t *db, size_t db_sz,
                                         size_t page_sz, uint32_t page_no,
                                         const char *table_name) {
  if (page_no == 0)
    return 0;
  size_t off = (size_t)(page_no - 1) * page_sz;
  if (off + page_sz > db_sz)
    return 0;

  const uint8_t *page = db + off;
  uint8_t type = page[0];
  int nc = (page[3] << 8) | page[4];

  if (type == PAGE_LEAF) {
    for (int c = 0; c < nc; c++) {
      uint16_t cell_off = (page[8 + 2 * c] << 8) | page[8 + 2 * c + 1];
      int vlen1;
      uint64_t payload_len = read_varint_scalar(page + cell_off, &vlen1);
      int vlen2;
      (void)read_varint_scalar(page + cell_off + vlen1, &vlen2);
      const uint8_t *payload = page + cell_off + vlen1 + vlen2;
      int hdrlen_bytes;
      uint64_t hdrlen = read_varint_scalar(payload, &hdrlen_bytes);
      size_t header_end = hdrlen;
      size_t hpos = (size_t)hdrlen_bytes;
      size_t dpos = header_end;
      char type_text[16] = {0}, name[128] = {0}, tbl[128] = {0};
      uint32_t root = 0;
      for (int col = 0; hpos < header_end; col++) {
        int slen;
        uint64_t st = read_varint_scalar(payload + hpos, &slen);
        hpos += (size_t)slen;
        if (st == 0)
          continue;
        if (st >= 13 && (st % 2)) {
          uint64_t len = (st - 13) / 2;
          if (dpos + len > payload_len)
            len = payload_len - dpos;
          if (col == 0)
            memcpy(type_text, payload + dpos, (len < 15 ? len : 15));
          if (col == 1)
            memcpy(name, payload + dpos, (len < 127 ? len : 127));
          if (col == 2)
            memcpy(tbl, payload + dpos, (len < 127 ? len : 127));
          dpos += (size_t)len;
        } else if (st >= 1 && st <= 6) {
          static const int blut[7] = {0, 1, 2, 3, 4, 6, 8};
          int blen = blut[st];
          if ((size_t)dpos + blen > payload_len)
            blen = (int)(payload_len - dpos);
          if (blen == 4 && col == 3) {
            root = (payload[dpos] << 24) | (payload[dpos + 1] << 16) |
                   (payload[dpos + 2] << 8) | payload[dpos + 3];
          }
          dpos += (size_t)blen;
        }
      }
      if (!strcmp(type_text, "table") && (!strcmp(name, table_name) ||
                                           !strcmp(tbl, table_name)))
        return root;
    }
  } else if (type == PAGE_INTERIOR) {
    for (int c = 0; c < nc; c++) {
      uint16_t cell_off = (page[12 + 2 * c] << 8) | page[12 + 2 * c + 1];
      uint32_t child = (page[cell_off] << 24) | (page[cell_off + 1] << 16) |
                       (page[cell_off + 2] << 8) | (page[cell_off + 3]);
      uint32_t res =
          find_table_rootpage_page(db, db_sz, page_sz, child, table_name);
      if (res)
        return res;
    }
    uint32_t right = (page[8] << 24) | (page[9] << 16) | (page[10] << 8) |
                     (page[11]);
    return find_table_rootpage_page(db, db_sz, page_sz, right, table_name);
  }

  return 0;
}

static uint32_t find_table_rootpage(const uint8_t *db, size_t db_sz,
                                    size_t page_sz, const char *table_name) {
  return find_table_rootpage_page(db, db_sz, page_sz, 1, table_name);
}

static void cpu_write_csv(const uint8_t *db, size_t page_sz, RecordTask *tasks,
                          int n_tasks, char *out, uint64_t *prefix) {
#pragma omp parallel for schedule(static, 16)
  for (int i = 0; i < n_tasks; i++) {
    const RecordTask *t = &tasks[i];
    const uint8_t *payload = db + (size_t)t->page * page_sz + t->offset;
    uint64_t start = prefix[i];
    char *dst = out + start;
    dst = u64toa(t->row_id, dst);
    *dst++ = ',';
    dst = u64toa(t->rowid_val, dst);
    *dst++ = ',';
    dst = decode_record(payload, (size_t)t->length, t->rowid_val, dst);
  }
}

static uint64_t *prefix_sum(uint32_t *lens, int n) {
  uint64_t *pref = (uint64_t *)malloc((size_t)(n + 1) * sizeof(uint64_t));
  pref[0] = 0;
  for (int i = 1; i <= n; i++)
    pref[i] = pref[i - 1] + (uint64_t)lens[i - 1];
  return pref;
}

int main(int argc, char **argv) {
  if (argc < 3) {
    fprintf(
        stderr,
        "usage: %s [--cuda] [--simd|--nosimd] [--table <name>] in.db out.csv\n",
        argv[0]);
    return 1;
  }

  int use_cuda = 0;
  const char *table_name = NULL;
  const char *in = NULL;
  const char *outp = NULL;

  for (int i = 1; i < argc; i++) {
    if (!strcmp(argv[i], "--cuda"))
      use_cuda = 1;
    else if (!strcmp(argv[i], "--simd"))
      use_simd_runtime = 1;
    else if (!strcmp(argv[i], "--nosimd"))
      use_simd_runtime = 0;
    else if (!strcmp(argv[i], "--table") && i + 1 < argc)
      table_name = argv[++i];
    else if (!in)
      in = argv[i];
    else if (!outp)
      outp = argv[i];
  }

  if (!in || !outp) {
    fprintf(stderr, "missing in/out\n");
    return 1;
  }

  size_t db_sz;
  uint8_t *db = mmap_file(in, &db_sz);
  size_t page_sz = detect_page_size(db);
  size_t n_pages = db_sz / page_sz;

  fprintf(stderr, "[+] pages=%zu size=%zu use_cuda=%d simd=%d\n", n_pages,
          page_sz, use_cuda, use_simd_runtime);

  uint32_t rootpage = 0;
  if (table_name) {
    rootpage = find_table_rootpage(db, db_sz, page_sz, table_name);
    if (!rootpage) {
      fprintf(stderr, "[!] table not found\n");
      return 1;
    }
    fprintf(stderr, "[+] table '%s' rootpage=%u\n", table_name, rootpage);
  }

  PageList leafs;
  pagelist_init(&leafs);
  if (table_name)
    collect_table_pages(db, db_sz, page_sz, rootpage, &leafs);
  else
    for (uint32_t i = 1; i < n_pages; i++)
      pagelist_push(&leafs, i);

  fprintf(stderr, "[+] collected %d leaf pages\n", leafs.count);

  RecordTask *tasks =
      (RecordTask *)malloc(leafs.count * MAX_CELLS * sizeof(RecordTask));
  int n_tasks = 0;

#ifdef __HIPCC__
  if (use_cuda) {
    RecordTask *d_tasks;
    int *d_total;
    int zero = 0;
    hipMalloc(&d_tasks, leafs.count * MAX_CELLS * sizeof(RecordTask));
    hipMalloc(&d_total, sizeof(int));
    hipMemcpy(d_total, &zero, sizeof(int), hipMemcpyHostToDevice);
    uint8_t *d_db;
    hipMalloc(&d_db, db_sz);
    hipMemcpy(d_db, db, db_sz, hipMemcpyHostToDevice);
    for (int p = 0; p < leafs.count; p++)
      gpu_scan_pages<<<1, 64>>>(d_db, page_sz, leafs.pages[p] + 1, d_tasks,
                                d_total);
    hipDeviceSynchronize();
    hipMemcpy(&n_tasks, d_total, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tasks, d_tasks, n_tasks * sizeof(RecordTask),
               hipMemcpyDeviceToHost);
    hipFree(d_tasks);
    hipFree(d_total);
    hipFree(d_db);
    fprintf(stderr, "[+] GPU decoded %d tasks\n", n_tasks);
  } else
#endif
  {
#pragma omp parallel for schedule(static, 8) reduction(+ : n_tasks)
    for (int pi = 0; pi < leafs.count; pi++) {
      uint32_t pg = leafs.pages[pi];
      const uint8_t *page = db + pg * page_sz;
      int nc = (page[3] << 8) | page[4];
      for (int c = 0; c < nc && c < MAX_CELLS; c++) {
        uint16_t off = (page[8 + 2 * c] << 8) | page[8 + 2 * c + 1];
        int vlen1;
        uint64_t pay = read_varint_auto(page + off, &vlen1);
        int vlen2;
        uint64_t rowid_val = read_varint_auto(page + off + vlen1, &vlen2);
        RecordTask t;
        t.row_id = ((uint64_t)pg << 32) | c;
        t.rowid_val = rowid_val;
        t.page = pg;
        t.offset = (uint16_t)(off + vlen1 + vlen2);
        t.length = (uint16_t)pay;
        tasks[pi * MAX_CELLS + c] = t;
      }
      n_tasks += nc;
    }
    fprintf(stderr, "[+] CPU parsed %d tasks\n", n_tasks);
  }

  uint32_t *lens = (uint32_t *)malloc(n_tasks * sizeof(uint32_t));
  for (int i = 0; i < n_tasks; i++)
    lens[i] = (uint32_t)(tasks[i].length + 128);
  uint64_t *pref = prefix_sum(lens, n_tasks);
  uint64_t total_bytes = pref[n_tasks];
  fprintf(stderr, "[+] writing %lu bytes output\n", total_bytes);

  int fd_out;
  char *outbuf = (char *)mmap_outfile(outp, total_bytes, &fd_out);
  cpu_write_csv(db, page_sz, tasks, n_tasks, outbuf, pref);
  msync(outbuf, total_bytes, MS_SYNC);
  munmap(outbuf, total_bytes);
  close(fd_out);
  munmap(db, db_sz);
  free(tasks);
  free(lens);
  free(pref);
  free(leafs.pages);
  fprintf(stderr, "[+] done.\n");
  return 0;
}
